#include "hip/hip_runtime.h"

// ----------------------------------------------------------------------------------
// Copyright 2016-2017 Michael-Angelo Yick-Hang Lam
//
// The development of this software was supported by the National Science Foundation
// (NSF) Grant Number DMS-1211713.
//
// This file is part of GADIT.
//
// GADIT is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License version 3 as published by
// the Free Software Foundation.
//
// GADIT is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with GADIT.  If not, see <http://www.gnu.org/licenses/>.
// ----------------------------------------------------------------------------------

// ----------------------------------------------------------------------------------
// Name:			main.cu
// Version: 		1.0
// Purpose:			Minimal example code of how to execute GADIT.
// ----------------------------------------------------------------------------------

#include "gadit_solver.h"

int main()
{

	// Allow switching between float and double precision.
	// Note: May remove in future versions and assume double precision.
	typedef double PRECISION;

	//select model ID and initial condition ID
	// see solver_template.h for list of values
	model::id const MODEL_ID = model::DEFAULT;
	initial_condition::id const IC_ID = initial_condition::LINEAR_WAVES;

	// select boundary conditions
	// NOTE: For now only symmetric boundary conditions are implemented 
	//       i.e. h_x=h_xxx=0. Will upload revision to the code in the following
	//       months that allow a cleaner implementation of multiple boundary condition.
	//       You may alter boundary_conditions.h and implement your own boundary conditions
	boundary_condtion_type const BC_X0 = boundary_condtion_type::SYMMETRIC;
	boundary_condtion_type const BC_Y0 = boundary_condtion_type::SYMMETRIC;
	boundary_condtion_type const BC_XN = boundary_condtion_type::SYMMETRIC;
	boundary_condtion_type const BC_YM = boundary_condtion_type::SYMMETRIC;

	// simplifying class reference
	typedef gadit_solver<PRECISION, MODEL_ID, IC_ID, BC_X0, BC_XN, BC_Y0, BC_YM> gadit_solver;

	// File contain all parameters that can be altered by the user. 
	parameters<PRECISION, MODEL_ID, IC_ID> paras;

	// Spatial partition parameters
	paras.spatial.ds = 0.05;
	paras.spatial.x0 = 0.0;
	paras.spatial.n = 519;
	paras.spatial.y0 = 0.0;
	paras.spatial.m = 519;

	// Model Parameters
	paras.model.cC = 0.0857;
	paras.model.cN = 1.67;
	paras.model.cK = 36.0;
	paras.model.b = 0.01;
	paras.model.beta = 1.0;
	paras.model.w = 0.05;

	// Parameters for initial condition
	paras.initial.h0 = 0.24;
	paras.initial.epx = 0.01;
	paras.initial.nx = 2 * 1;
	paras.initial.epy = 0.01;
	paras.initial.ny = 2 * 1;

	// Temporal parameters
	paras.temporal.t_start  = 0.0;
	paras.temporal.dt_out	= 1.0;
	paras.temporal.t_end    = paras.temporal.dt_out * 500;

	// backup time for solution in minutes 
	paras.backup.updateTime = 5;

	// Add '/' to end if not using execution directory as root e.g. some_folder/
	paras.io.root_directory = "";

	// Toggle to control output of status of GADIT solver
	paras.io.is_console_output = true;
	paras.io.is_full_text_output = false;


	// It is not necessary the change the remaining parameters,
	// but feel free to do so.
	paras.newton.error_tolerence = pow(10, -10);

	// Testing shows 10 produces best effective time step
	// i.e. dt/interation_count
	paras.newton.max_iterations = 10;
	// Applies a minimum amount iterations with out convergence checks
	paras.newton.min_iterations = 3;

	paras.temporal.dt_min = pow(10, -13);
	paras.temporal.dt_max = 0.1*paras.temporal.dt_out;

	// set large to prevent excessive dt increase
	// that with results in immediate newton convergence
	// failure within thresholds.
	paras.temporal.min_stable_step = 500;
	// dt is allowed to increase exponentially once min_step is
	// reach. After failure,  min_stable_steps much be achieved
	// before dt can be increased again.
	paras.temporal.dt_ratio_increase = 1.07;
	paras.temporal.dt_ratio_decrease = 1.05;

	// setting to a very small will only affect the
	// start up of GADIT. GADIT allows exponential growth
	// of the time step
	paras.temporal.dt_init = 0.000001*paras.temporal.dt_out;

	// initializes solver and evolve solution
	gadit_solver *solver;
	solver = new gadit_solver();

	solver->initialize(paras);
	solver->solve_model_evolution();

	return 0;
}

